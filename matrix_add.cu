
#include <hip/hip_runtime.h>
// GPU
__global__ void kernel_1t1e(float *A, float *B, float *C, unsigned long WIDTH) {
	// To DO: Device a row major indexing
	int rowID = threadIdx.y + blockIdx.y * blockDim.y; 	// Row address
	int colID = threadIdx.x + blockIdx.x * blockDim.x;	// Column Address
	int elemID;											// Element address

    // a_ij = a[i][j], where a is in row major order
	if(rowID < WIDTH && colID < WIDTH){
		elemID = colID + rowID * WIDTH; 				
		C[elemID] = A[elemID] + B[elemID];
	}
}

__global__ void kernel_1t1r(float *A, float *B, float *C, unsigned long WIDTH) {
	// To DO: Each thread = 1 output row
	int rowID = threadIdx.y + blockIdx.y * blockDim.y;	// Row address

	if(rowID < WIDTH) {
		for(int i = 0; i<WIDTH; i++){
			//elemID = colID + rowID * WIDTH; 
			C[i + rowID*WIDTH] = A[i + rowID*WIDTH] + B[i + rowID*WIDTH];
		}
	}
}
